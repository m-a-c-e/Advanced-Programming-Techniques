/*
 * Author: Manan Patel
 * Class: ECE 6122
 * Last Date Modified: 15/11/2021
 *
 * Description:
 * Heat distribution on a thin plate.
 * Uses Laplace's finite difference method to calculte the 
 * approximate distribution of thin plate within a given number of iterations.
 */


// imports
#include <iostream>
#include <stdio.h>
#include <math.h>
#include <fstream>
#include <sstream>
#include <hip/hip_runtime.h>

using namespace std;

__global__ void compute_step(double* arr_list, double* updated_arr_list, int n_total)
{
/* Purpose: To compute the temperature distribution within a thin
*  plate using laplace finite difference method.
*  Args:
		arr_list -> array which holds the current temperature distribution
		updated_arr_list -> array which holds the newly computed temperature distribution
		n_total -> n_total * n_total is the total length of the array 
*  Returns:
		NA
*/
	int thId = threadIdx.x;
	int stride = blockDim.x;

	int total = n_total * n_total;
	for (int i = thId; i < total; i+=stride)
	{
		if(i <= n_total || (i + 1) % n_total == 0 || (i % n_total == 0) || (i >= total - n_total - 1))
		{
			updated_arr_list[i] = arr_list[i];
			continue;
		}
		updated_arr_list[i] = 0.25 * (arr_list[i - 1] +
			arr_list[i + 1] +
			arr_list[i - n_total] +
			arr_list[i + n_total]);
	}
}

__global__ void update_step(double* arr_list, double* updated_arr_list, int n_total)
{
/* Purpose: To set the current temperature distribution to
*  the computed one.
*  Args:
		arr_list -> array which holds the current temperature distribution
		updated_arr_list -> array which holds the newly computed temperature distribution
		n_total -> n_total * n_total is the total length of the array 
*  Returns:
		NA
*/
	int thId = threadIdx.x;
	int stride = blockDim.x;

	int total = n_total * n_total;
	for (int i = thId; i < total; i+=stride)
	{
		arr_list[i] = updated_arr_list[i];
	}
}

bool checkIfInt(string str)
{
/* Purpose: To check if the input string is an integer
*  Args:
		str -> string	
*  Returns:
		true or false
*/

	for(int i = 0; i < str.length(); i ++)
	{
		if(isdigit(str[i]) == false)
		{
			return false;
		}
	}
	return true;
}


int main(int argc, char* argv[])
{
	// Start to measure time
	hipEvent_t start;
	hipEvent_t stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);

	// Checking for commmand line arguments
	if(argc != 5)
	{
		cout << "Invalid parameters. Please check your values." << endl;
		return 0;
	}

	string n_str = "-N";
	string iterations_str = "-l";
	double n = -1;
	double iterations = -1;


	for(int i = 0; i < argc; i ++)
	{
		if(n_str.compare(argv[i]) == 0)
		{
			// if -N input is non-numeric
			if(!checkIfInt(argv[i+1]))
			{
				cout << "Invalid parameters. Please check your values." << endl;
				return 0;
			}
			n = strtod(argv[i + 1], nullptr);
		}	
		if(iterations_str.compare(argv[i]) == 0)
		{
			// if -l input is non-numeric
			if(!checkIfInt(argv[i+1]))
			{
				cout << "Invalid parameters. Please check your values." << endl;
				return 0;
			}
			iterations = strtod(argv[i + 1], nullptr);
		}
	}

	// If n or iterations is negative
	if(n <= 0 || iterations <= 0)
	{
		cout << "Invalid parameters. Please check your values." << endl;
		return 0;
	}


	double n_total = n + 2;			// includes the points which are not to be updated as well

	// allocate memory and set initial temperature dist
	double* arr_list = (double*)malloc(sizeof(double) * n_total * n_total);
	for (int i = 0; i < n_total * n_total; i++)
	{
		if (i > (0.3 * (n_total - 1)) && i < (0.7 * (n_total - 1)))
		{
			arr_list[i] = 100.0;
		}
		else
		{
			arr_list[i] = 20.0;
		}
	}

	// Make memory on device and copy initial array
	double* d_arr_list;
	hipMalloc((void**)&d_arr_list, sizeof(double) * n_total * n_total);
	hipMemcpy(d_arr_list, arr_list, sizeof(double) * n_total * n_total, hipMemcpyHostToDevice);

	double* d_updated_arr_list;
	hipMalloc((void**)&d_updated_arr_list, sizeof(double) * n_total * n_total);

	// check the max number of threads allowed per block
	// set the number of threads to launch
	hipSetDevice(0);
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, 0);
	int maxThreads = deviceProp.maxThreadsPerBlock;
	int n_threads = 0;
	if(maxThreads > n_total * n_total)
	{
		n_threads = n_total * n_total;
	}
	else
	{
		n_threads = maxThreads;
	}

	// Run the kernels iterations number of times
	for (int k = 0; k < iterations; k++)
	{
		// computes the temp dist
		compute_step<<<1, n_threads>>>(d_arr_list, d_updated_arr_list, n_total);
		hipDeviceSynchronize();

		// sets the temp dist from d_updated_arr_list to d_arr_list
		update_step<<<1, n_threads>>>(d_arr_list, d_updated_arr_list, n_total);
		hipDeviceSynchronize();
	}
	
	// copy array back to host
	hipMemcpy(arr_list, d_arr_list, sizeof(double) * n_total * n_total, hipMemcpyDeviceToHost);

	// writing to a file
	ofstream myfile;
	myfile.open("Temperatures.csv", ios::binary);
	for (int i = 0; i < n_total * n_total; i+=n_total)
	{
		stringstream ss;
		for (int j = 0; j < n_total; j++)
		{
			ss << arr_list[i + j] << ',';
		}
		ss << '\n';
		myfile << ss.str();
	}

	// Deallocate memory on device and host
	free(arr_list);
	hipFree(d_arr_list);
	hipFree(d_updated_arr_list);
	myfile.close();

	// Output total time taken by the program
	cout.precision(5);
	hipEventRecord(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	cout << milliseconds << endl;

	return 0;
}
